/**
* Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/***********************************************************************************\
* Standard Includes
\***********************************************************************************/
#include <iostream>
#include <string>
#include <assert.h>
/* For the CUDA runtime routines (prefixed with "cuda_") */
#include <hip/hip_runtime.h>

/***********************************************************************************\
* # Defines
\***********************************************************************************/
#define TRUE (1)
#define FALSE (0)

/* Number of rows and columns of the global memory */
#define NUM_OF_GLOBAL_ROWS (16000)
#define NUM_OF_GLOBAL_COLS (16000)

/* Number of threads in each block */
#define BLOCK_SIZE (32)

/* Convolution Kernel size */
#define KERNEL_SIZE (2)

/* Number of row and columns of the local memory */
#define NUM_OF_LOCAL_ROWS (BLOCK_SIZE + (2 * KERNEL_SIZE))
#define NUM_OF_LOCAL_COLS (BLOCK_SIZE + (2 * KERNEL_SIZE))

/***********************************************************************************\
* Enums
\***********************************************************************************/
typedef enum Status_Tag
{
	PASSED,
	FAILED
}Status_T;

typedef enum Cuda_Event_Tag
{
	ALLOCATE_DEVICE_MATRIX_A,
	ALLOCATE_DEVICE_MATRIX_B,
	COPY_MATRIX_A_FROM_HOST_TO_DEVICE,
	LAUNCH_KERNEL_CONV2DDEVICE,
	DEVICE_SYNCHRONIZATION,
	COPY_MATRIX_B_FROM_DEVICE_TO_HOST,
	FREE_DEVICE_MATRIX_A,
	FREE_DEVICE_MATRIX_B
}Cuda_Event_T;

typedef enum Corner_Cell_Name_Tag
{
	LEFT_TOP_PADDING_CORNER,
	RIGHT_TOP_PADDING_CORNER,
	LEFT_BOTTOM_PADDING_CORNER,
	RIGHT_BOTTOM_PADDING_CORNER,
	NUM_OF_CORNERS
}Corner_Cell_Name_T;

typedef enum Ver_Side_Cell_Name_Tag
{
	LEFT_PADDING_CELL,
	RIGHT_PADDING_CELL,
	NUM_OF_VER_SIDES
}Ver_Side_Cell_Name_T;

typedef enum Hor_Side_Cell_Name_Tag
{
	TOP_PADDING_CELL,
	BOTTOM_PADDING_CELL,
	NUM_OF_HOR_SIDES
}Hor_Side_Cell_Name_T;

/***********************************************************************************\
* Structures
\***********************************************************************************/
typedef struct Result_Tag
{
	Status_T status;
	int index;
}Result_T;

typedef struct Cell_Tag
{
	int r_idx; /* Row index */
	int c_idx; /* Column index */
}Cell_T;

/***********************************************************************************\
* Function Macros
\***********************************************************************************/
#define MATRIX_TO_ARRAY_INDEX(r_idx, c_idx, num_cols) ((r_idx*num_cols) + c_idx)

/***********************************************************************************\
* CUDA Kernel Device code for 2D Convolution
\***********************************************************************************/
__global__ void
conv2DDevice(const int *in,  int *out)
{
	int g_col_idx = blockDim.x * blockIdx.x + threadIdx.x;
	int g_row_idx = blockDim.y * blockIdx.y + threadIdx.y;
	int l_col_idx = threadIdx.x + KERNEL_SIZE;
	int l_row_idx = threadIdx.y + KERNEL_SIZE;

	__shared__ int local[NUM_OF_LOCAL_ROWS*NUM_OF_LOCAL_COLS];

	/* Convert from matrix indexing to array indexing */
	int g_idx = MATRIX_TO_ARRAY_INDEX(g_row_idx, g_col_idx, NUM_OF_GLOBAL_COLS);
	int l_idx = MATRIX_TO_ARRAY_INDEX(l_row_idx, l_col_idx, NUM_OF_LOCAL_COLS);

	if ((g_row_idx < NUM_OF_GLOBAL_ROWS) && (g_col_idx < NUM_OF_GLOBAL_COLS))
	{
		/* Read input elements into shared memory */
		
		/* Fill the internal (BLOCK_SIZE*BLOCK_SIZE) matrix */
		local[l_idx] = in[g_idx];

		/* Fill the left and right padding columns of local memory */
		if (threadIdx.x < KERNEL_SIZE)
		{
			Cell_T l_ver_side[NUM_OF_VER_SIDES];
			Cell_T g_ver_side[NUM_OF_VER_SIDES];

			/* Find left and right padding column indices of local memory */
			l_ver_side[LEFT_PADDING_CELL].r_idx = l_row_idx;
			l_ver_side[LEFT_PADDING_CELL].c_idx = l_col_idx - KERNEL_SIZE;

			l_ver_side[RIGHT_PADDING_CELL].r_idx = l_row_idx;
			l_ver_side[RIGHT_PADDING_CELL].c_idx = l_col_idx + BLOCK_SIZE;

			/* Find indices of global memory whose data needs to be filled 
			   into the left and right padding columns of local memory */
			g_ver_side[LEFT_PADDING_CELL].r_idx = g_row_idx;
			g_ver_side[LEFT_PADDING_CELL].c_idx = g_col_idx - KERNEL_SIZE;

			g_ver_side[RIGHT_PADDING_CELL].r_idx = g_row_idx;
			g_ver_side[RIGHT_PADDING_CELL].c_idx = g_col_idx + BLOCK_SIZE;

			for (int cell = LEFT_PADDING_CELL; cell < NUM_OF_VER_SIDES; ++cell)
			{
				bool within_bounds = FALSE;

				/* Check if the cell is within bounds of global matrix */
				if (LEFT_PADDING_CELL == cell) {
					within_bounds = (g_ver_side[cell].c_idx >= 0);
				}

				if (RIGHT_PADDING_CELL == cell) {
					within_bounds = (g_ver_side[cell].c_idx < NUM_OF_GLOBAL_COLS);
				}

				/* Copy corner into local memory if it is within the bounds of global matrix */
				/* Convert from matrix indexing to array indexing */
				int pad_l_idx = MATRIX_TO_ARRAY_INDEX(l_ver_side[cell].r_idx, l_ver_side[cell].c_idx, NUM_OF_LOCAL_COLS);
				int pad_g_idx = MATRIX_TO_ARRAY_INDEX(g_ver_side[cell].r_idx, g_ver_side[cell].c_idx, NUM_OF_GLOBAL_COLS);
				if (TRUE == within_bounds) {
					local[pad_l_idx] = in[pad_g_idx];
				}
				else {
					local[pad_l_idx] = 0;
				}
			}
		}

		/* Fill the top and bottom padding rows */
		if (threadIdx.y < KERNEL_SIZE)
		{
			Cell_T l_hor_side[NUM_OF_HOR_SIDES];
			Cell_T g_hor_side[NUM_OF_HOR_SIDES];

			/* Find top and bottom padding row indices of local memory */
			l_hor_side[TOP_PADDING_CELL].r_idx = l_row_idx - KERNEL_SIZE;
			l_hor_side[TOP_PADDING_CELL].c_idx = l_col_idx;

			l_hor_side[BOTTOM_PADDING_CELL].r_idx = l_row_idx + BLOCK_SIZE;
			l_hor_side[BOTTOM_PADDING_CELL].c_idx = l_col_idx;

			/* Find indices of global memory whose data needs to be filled 
			   into the top and bottom padding rows of local memory */
			g_hor_side[TOP_PADDING_CELL].r_idx = g_row_idx - KERNEL_SIZE;
			g_hor_side[TOP_PADDING_CELL].c_idx = g_col_idx;

			g_hor_side[BOTTOM_PADDING_CELL].r_idx = g_row_idx + BLOCK_SIZE;
			g_hor_side[BOTTOM_PADDING_CELL].c_idx = g_col_idx;

			for (int cell = TOP_PADDING_CELL; cell < NUM_OF_HOR_SIDES; ++cell)
			{
				bool within_bounds = FALSE;

				/* Check if the cell is within bounds of global matrix */
				if (TOP_PADDING_CELL == cell) {
					within_bounds = (g_hor_side[cell].r_idx >= 0);
				}

				if (BOTTOM_PADDING_CELL == cell) {
					within_bounds = (g_hor_side[cell].r_idx < NUM_OF_GLOBAL_ROWS);
				}

				/* Copy corner into local memory if it is within the bounds of global matrix */
				/* Convert from matrix indexing to array indexing */
				int pad_l_idx = MATRIX_TO_ARRAY_INDEX(l_hor_side[cell].r_idx, l_hor_side[cell].c_idx, NUM_OF_LOCAL_COLS);
				int pad_g_idx = MATRIX_TO_ARRAY_INDEX(g_hor_side[cell].r_idx, g_hor_side[cell].c_idx, NUM_OF_GLOBAL_COLS);
				if (TRUE == within_bounds) {
					local[pad_l_idx] = in[pad_g_idx];
				}
				else {
					local[pad_l_idx] = 0;
				}
			}
		}

		/* Fill the corners */
		if ((threadIdx.x) < KERNEL_SIZE && (threadIdx.y < KERNEL_SIZE))
		{
			Cell_T l_corner[NUM_OF_CORNERS];
			Cell_T g_corner[NUM_OF_CORNERS];

			/* Find left top, right top, left bottom and right bottom padding
			   corner indices of local memory */
			l_corner[LEFT_TOP_PADDING_CORNER].r_idx = l_row_idx - KERNEL_SIZE;
			l_corner[LEFT_TOP_PADDING_CORNER].c_idx = l_col_idx - KERNEL_SIZE;

			l_corner[RIGHT_TOP_PADDING_CORNER].r_idx = l_row_idx - KERNEL_SIZE;
			l_corner[RIGHT_TOP_PADDING_CORNER].c_idx = l_col_idx + BLOCK_SIZE;

			l_corner[LEFT_BOTTOM_PADDING_CORNER].r_idx = l_row_idx + BLOCK_SIZE;
			l_corner[LEFT_BOTTOM_PADDING_CORNER].c_idx = l_col_idx - KERNEL_SIZE;

			l_corner[RIGHT_BOTTOM_PADDING_CORNER].r_idx = l_row_idx + BLOCK_SIZE;
			l_corner[RIGHT_BOTTOM_PADDING_CORNER].c_idx = l_col_idx + BLOCK_SIZE;

			/* Find indices of global memory whose data needs to be filled 
			   into the left top, right top, left bottom and right bottom padding
			   corners of local memory */
			g_corner[LEFT_TOP_PADDING_CORNER].r_idx = g_row_idx - KERNEL_SIZE;
			g_corner[LEFT_TOP_PADDING_CORNER].c_idx = g_col_idx - KERNEL_SIZE;

			g_corner[RIGHT_TOP_PADDING_CORNER].r_idx = g_row_idx - KERNEL_SIZE;
			g_corner[RIGHT_TOP_PADDING_CORNER].c_idx = g_col_idx + BLOCK_SIZE;

			g_corner[LEFT_BOTTOM_PADDING_CORNER].r_idx = g_row_idx + BLOCK_SIZE;
			g_corner[LEFT_BOTTOM_PADDING_CORNER].c_idx = g_col_idx - KERNEL_SIZE;

			g_corner[RIGHT_BOTTOM_PADDING_CORNER].r_idx = g_row_idx + BLOCK_SIZE;
			g_corner[RIGHT_BOTTOM_PADDING_CORNER].c_idx = g_col_idx + BLOCK_SIZE;

			for (int corner = LEFT_TOP_PADDING_CORNER; corner < NUM_OF_CORNERS; ++corner)
			{
				bool within_bounds = FALSE;

				/* Check if the corner is within bounds of global matrix */
				if (LEFT_TOP_PADDING_CORNER == corner){
					within_bounds = ((g_corner[corner].r_idx >= 0) && (g_corner[corner].c_idx >= 0));
				}

				if (RIGHT_TOP_PADDING_CORNER == corner){
					within_bounds = ((g_corner[corner].r_idx >= 0) && (g_corner[corner].c_idx < NUM_OF_GLOBAL_COLS));
				}

				if (LEFT_BOTTOM_PADDING_CORNER == corner){
					within_bounds = ((g_corner[corner].r_idx < NUM_OF_GLOBAL_ROWS) && (g_corner[corner].c_idx >= 0));
				}

				if (RIGHT_BOTTOM_PADDING_CORNER == corner){
					within_bounds = ((g_corner[corner].r_idx < NUM_OF_GLOBAL_ROWS) && (g_corner[corner].c_idx < NUM_OF_GLOBAL_COLS));
				}

				/* Copy corner into local memory if it is within the bounds of global matrix */
				/* Convert from matrix indexing to array indexing */
				int pad_l_idx = MATRIX_TO_ARRAY_INDEX(l_corner[corner].r_idx, l_corner[corner].c_idx, NUM_OF_LOCAL_COLS);
				int pad_g_idx = MATRIX_TO_ARRAY_INDEX(g_corner[corner].r_idx, g_corner[corner].c_idx, NUM_OF_GLOBAL_COLS);
				if (TRUE == within_bounds){
					local[pad_l_idx] = in[pad_g_idx];
				}
				else {
					local[pad_l_idx] = 0;
				}
			}
		}

		__syncthreads();

		/* Apply convolution */
		int result = 0;
		for (int row_offset = -KERNEL_SIZE; row_offset <= KERNEL_SIZE; ++row_offset)
		{
			for (int col_offset = -KERNEL_SIZE; col_offset <= KERNEL_SIZE; ++col_offset)
			{
				/* Convert local matrix row and column to local element index */
				int l_ele_idx = MATRIX_TO_ARRAY_INDEX((l_row_idx + row_offset), (l_col_idx + col_offset), NUM_OF_LOCAL_COLS);
				result += local[l_ele_idx];
			}
		}

		/* Store the result */
		out[g_idx] = result;
	}
}

/***********************************************************************************\
* Host code for 2D Convolution and comparing the result with device 2D Convolution
\***********************************************************************************/
Result_T checkResult(int* h_A, int* h_B)
{
	Result_T result;
	result.status = PASSED;
	result.index = -1;

	int mat_size = NUM_OF_GLOBAL_ROWS * NUM_OF_GLOBAL_COLS;
	
	for (int ele_idx = 0; ele_idx < mat_size; ++ele_idx)
	{
		/* Convert input element index to input matrix row and column */
		int mat_row_num = ele_idx / NUM_OF_GLOBAL_COLS;
		int mat_col_num = ele_idx % NUM_OF_GLOBAL_COLS;

		int sum = 0;

		for (int row_offset = -KERNEL_SIZE; row_offset <= KERNEL_SIZE; ++row_offset)
		{
			for (int col_offset = -KERNEL_SIZE; col_offset <= KERNEL_SIZE; ++col_offset)
			{
				/* Get kernel matrix row and column */
				int mat_ker_row_num = mat_row_num + row_offset;
				int mat_ker_col_num = mat_col_num + col_offset;

				if ((mat_ker_row_num >= 0) && (mat_ker_row_num < NUM_OF_GLOBAL_ROWS) &&
					(mat_ker_col_num >= 0) && (mat_ker_col_num < NUM_OF_GLOBAL_COLS))
				{
					/* Convert kernel matrix row and column to kernel element index */
					int ker_ele_idx = MATRIX_TO_ARRAY_INDEX(mat_ker_row_num, mat_ker_col_num, NUM_OF_GLOBAL_COLS);

					if (ker_ele_idx >= 0)
					{
						sum += h_A[ker_ele_idx];
					}
				}
			}
		}

		if (h_B[ele_idx] != sum) {
			result.status = FAILED;
			result.index = ele_idx;
			return result;
		}
	}

	return result;
}

/***********************************************************************************\
* Host code to initialize input matrix
\***********************************************************************************/
void initHostInputMatrix(int* h_A)
{
	for (int idx = 0; idx < (NUM_OF_GLOBAL_ROWS*NUM_OF_GLOBAL_COLS); ++idx)
	{
		h_A[idx] = (idx / NUM_OF_GLOBAL_COLS) + 1;
	}
}

/***********************************************************************************\
* Function to check CUDA error
\***********************************************************************************/
inline hipError_t checkCuda(hipError_t result, Cuda_Event_T cuda_event)
{
	char error_string[100];

	switch (cuda_event)
	{
	case ALLOCATE_DEVICE_MATRIX_A:
		strcpy(error_string, "Failed to allocate device matrix A");
		break;
	case ALLOCATE_DEVICE_MATRIX_B:
		strcpy(error_string, "Failed to allocate device matrix B");
		break;
	case COPY_MATRIX_A_FROM_HOST_TO_DEVICE:
		strcpy(error_string, "Failed to copy matrix A from host to device");
		break;
	case LAUNCH_KERNEL_CONV2DDEVICE:
		strcpy(error_string, "Failed to launch conv2DDevice kernel");
		break;
	case DEVICE_SYNCHRONIZATION:
		strcpy(error_string, "Failed to synchronize");
		break;
	case COPY_MATRIX_B_FROM_DEVICE_TO_HOST:
		strcpy(error_string, "Failed to copy matrix B from device to host");
		break;
	case FREE_DEVICE_MATRIX_A:
		strcpy(error_string, "Failed to free device matrix A");
		break;
	case FREE_DEVICE_MATRIX_B:
		strcpy(error_string, "Failed to free device matrix B");
		break;
	default:
		strcpy(error_string, "NOT DUE TO ONE OF THE CUDA EVENTS");
		break;
	}

	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %s (error code: %s)\n", error_string, hipGetErrorString(result));
		assert(result == hipSuccess);
		exit(EXIT_FAILURE);
	}

	return result;
}

/***********************************************************************************\
* Host main routine
\***********************************************************************************/
int main(void)
{
	/* Print the matrix dimension to be used, and compute its size */
	int numElements = NUM_OF_GLOBAL_ROWS * NUM_OF_GLOBAL_COLS;
	size_t size = numElements * sizeof(int);
	printf("[Convolution of matrix of (%d, %d) elements]\n", NUM_OF_GLOBAL_ROWS, NUM_OF_GLOBAL_COLS);

	/* Allocate the host input matrix A */
	int *h_A = (int *)malloc(size);

	/* Allocate the host output matrix B */
	int *h_B = (int *)malloc(size);

	/* Verify that allocations succeeded */
	if (h_A == NULL || h_B == NULL)
	{
		fprintf(stderr, "Failed to allocate host matrix!\n");
		exit(EXIT_FAILURE);
	}

	/* Initialize the host input matrix */
	initHostInputMatrix(h_A);

	/* Allocate the device input matrix A */
	int *d_A = NULL;
	checkCuda(hipMalloc((void **)&d_A, size), ALLOCATE_DEVICE_MATRIX_A);

	/* Allocate the device output matrix B */
	int *d_B = NULL;
	checkCuda(hipMalloc((void **)&d_B, size), ALLOCATE_DEVICE_MATRIX_B);

	/* Copy the host input matrix A in host memory to the device input matrix in
	   device memory */
	printf("Copy input data from the host memory to the CUDA device\n");
	checkCuda(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice), COPY_MATRIX_A_FROM_HOST_TO_DEVICE);

	/* Launch the 2D Convolution CUDA Kernel */
	dim3 block_dim(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid_dim(((NUM_OF_GLOBAL_COLS + BLOCK_SIZE - 1) / BLOCK_SIZE), ((NUM_OF_GLOBAL_ROWS + BLOCK_SIZE - 1) / BLOCK_SIZE));
	printf("CUDA kernel launch with (%d,%d) blocks of (%d,%d) threads\n", grid_dim.x, grid_dim.y, block_dim.x, block_dim.y);
	conv2DDevice<<<grid_dim, block_dim>>>(d_A, d_B);
	checkCuda(hipGetLastError(), LAUNCH_KERNEL_CONV2DDEVICE);

	checkCuda(hipDeviceSynchronize(), DEVICE_SYNCHRONIZATION);

	/* Copy the device result vector in device memory to the host result vector
	   in host memory */
	printf("Copy output data from the CUDA device to the host memory\n");
	checkCuda(hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost), COPY_MATRIX_B_FROM_DEVICE_TO_HOST);

	/* Verify that the result vector is correct */
	Result_T result = checkResult(h_A, h_B);
	if(FAILED == result.status)
	{
		fprintf(stderr, "Result verification failed at element %d!\n", result.index);
		exit(EXIT_FAILURE);
	}

	printf("Test PASSED\n");

	/* Free device global memory */
	checkCuda(hipFree(d_A), FREE_DEVICE_MATRIX_A);
	checkCuda(hipFree(d_B), FREE_DEVICE_MATRIX_B);

	/* Free host memory */
	free(h_A);
	free(h_B);

	printf("Done\n");

	return 0;
}

